#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void vector_mul(int *a, int *b, int *c){
	int i, j, soma = 0;

	
	int coluna = blockIdx.x * blockDim.x + threadIdx.x;
	int linha =  blockIdx.y * blockDim.y + threadIdx.x;
	//int index = blockIdx.x * blockDim.x + threadIdx.x;

	for (i = 0; i < 2048; i++) {
		soma += a[(2048 * linha + i] * b[2048 * i + coluna];		
	}

	// ((sizeof(int)+N) * linha) + (sizeof(int) * coluna)
	c[(2048 * linha) + coluna] = soma;
}

#define N (2048*2048)
#define THREADS_PER_BLOCK 512

int main(){
	int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	int size = N * sizeof(int);

	hipMalloc((void **) &d_a, size);
	hipMalloc((void **) &d_b, size);
	hipMalloc((void **) &d_c, size);

	a = (int *)malloc(size);
	b = (int *)malloc(size);
	c = (int *)malloc(size);

	for(int i = 0; i < N; i++){
		a[i] = b[i] = i;
		c[i] = 0;
	}

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);


	dim3 tbloco = dim3(
	vector_mul<<< (N + (THREADS_PER_BLOCK-1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>(d_a, d_b, d_c);
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
	printf("c[0] = %d\n",c[0]);
	printf("c[%d] = %d\n", N-1, c[N-1]);

	free(a); free(b); free(c);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}
