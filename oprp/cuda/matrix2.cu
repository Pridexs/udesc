
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>

#define NLINHAS 2048
#define NCOLUNAS 2048
#define THREADS_PER_BLOCK 1024
//32x32
#define NTHREADS 32
#define NCOLUNASPERBLOCK 32
#define NLINHASPERBLOCK 32

__global__ void vector_mul(int *a, int *b, int *c) {
    int i, z, soma = 0;

    int coluna = blockIdx.x * blockDim.x + threadIdx.x;
    int linha =  blockIdx.y * blockDim.y + threadIdx.y;

    __shared__ int s_a[NLINHASPERBLOCK][NCOLUNASPERBLOCK];
    __shared__ int s_b[NLINHASPERBLOCK][NCOLUNASPERBLOCK];

	for (z = 0; z < gridDim.x; z++) {
        s_a[threadIdx.y][threadIdx.x] = a[ (NCOLUNAS * (blockIdx.y * NLINHASPERBLOCK + threadIdx.y)) + (z * NCOLUNASPERBLOCK + threadIdx.x) ];
        s_b[threadIdx.y][threadIdx.x] = b[ (NCOLUNAS * (z * NLINHASPERBLOCK + threadIdx.y)) + blockIdx.x * NCOLUNASPERBLOCK + threadIdx.x ];
        
        __syncthreads();

		for (i = 0; i < NLINHASPERBLOCK; i++) {
	    	soma += s_a[threadIdx.y][i] * s_b[i][threadIdx.x];
	    }

        __syncthreads();
	}

    //printf("%d %d\n", linha, coluna);
    c[linha * NLINHAS + coluna] = soma;
}

int main(){
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int size = NLINHAS * NCOLUNAS * sizeof(int);
    int i, j, n;

    struct timeval timevalA;
	struct timeval timevalB;

    hipMalloc((void **) &d_a, size);
    hipMalloc((void **) &d_b, size);
    hipMalloc((void **) &d_c, size);

    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(size);

    for(i = 0; i < NLINHAS*NCOLUNAS; i++){
//        a[i] = b[i] = i % 10;
        c[i] = 0;
    }

    scanf("%d", &n);

    for (int i = 0; i < NLINHAS; i++) {
        for (j = 0; j < NLINHAS; j++) {
            scanf("%d", &a[i * NLINHAS + j]);
        }
    }

    for (int i = 0; i < NLINHAS; i++) {
        for (j = 0; j < NLINHAS; j++) {
            scanf("%d", &b[i * NLINHAS + j]);
        }
    }

    gettimeofday(&timevalA,NULL);
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    dim3 tbloco = dim3(64, 64,1);
    dim3 tthreads = dim3(32, 32, 1);
    vector_mul<<<tbloco,tthreads>>>(d_a, d_b, d_c);
    
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
    gettimeofday(&timevalB,NULL);
    
    // imprimir primeira coluna
    // for (i = 0; i < NLINHAS; i++) {
    //     for (j = 0; j < NLINHAS; j++) {
    //         printf("%d ", c[i * NLINHAS + j]);
    //     }
    //     printf("\n");
    // }
    //printf("\n");

    printf("%.5lf\n", timevalB.tv_sec-timevalA.tv_sec+(timevalB.tv_usec-timevalA.tv_usec)/(double)1000000);

    free(a); free(b); free(c);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
