#include "hip/hip_runtime.h"
#include <stdio.h>

#define NLINHAS 4096
#define NCOLUNAS 4096
#define THREADS_PER_BLOCK 1024
#define

__global__ void vector_mul(int *a, int *b, int *c) {
    int i, j, z, soma = 0;

    int coluna = blockIdx.x * blockDim.x + threadIdx.x;
    int linha =  blockIdx.y * blockDim.y + threadIdx.y;

	// nColunasPerBlock should be equal to nLinhasPerBlock
	// for now at least
	int nColunasPerBlock = (NCOLUNAS/blockDim.x);
	int nLinhasPerBlock = (NLINHAS/blockDim.y);

    __shared__ int s_a[nLinhasPerBlock][nColunasPerBlock];
    __shared__ int s_b[nLinhasPerBlock][nColunasPerBlock];

    // carregar pra memoria compartilhada da GPU

	// blockDim.x DEVEM SER IGUAIS blockDim.y
    // Copia Matriz
	int nColunasPerBlock = (NCOLUNAS/blockDim.x);
	for (z = 0; z < blockDim.x; z++) {
		for (i = 0; i < nColunasPerBlock; i++) {
			s_a[ blockIdx.y * nLinhasPerBlock  ][ z * nColunasPerBlock + i ] = a
		}

		// hipMemcpy(&d_a[0], s_a,
	    // 	(NLINHAS/blockDim.y) * NCOLUNAS * sizeof(int), hipMemcpyDeviceToDevice);
		//
	    // // Copia Matriz B
	    // for(i = 0; i < NLINHAS; i++) {
	    //     hipMemcpy(&d_b[ (i * NCOLUNAS * sizeof(int)) + (blockDim.x * sizeof(int) * (NCOLUNAS/blockDim.x)) ],
	    //     s_b, sizeof(int) * (NCOLUNAS/blockDim.x), hipMemcpyDeviceToDevice);
	    // }


	    __syncthreads();

		for (i = 0; i < nLinhasPerBlock; i++) {
	    	soma += s_a[i] * s_b[i];
	    }
	}

    c[(NLINHAS * linha) + coluna] = soma;
}

int main(){
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int size = NLINHAS * NCOLUNAS * sizeof(int);

    hipMalloc((void **) &d_a, size);
    hipMalloc((void **) &d_b, size);
    hipMalloc((void **) &d_c, size);

    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(size);

    for(int i = 0; i < NLINHAS*NCOLUNAS; i++){
        a[i] = b[i] = i;
        c[i] = 0;
    }

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    dim3 tthreads = dim3(32, 32, 1);
    dim3 tbloco = dim3(1,1,1);
    //vector_mul<<< (N + (THREADS_PER_BLOCK-1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>(d_a, d_b, d_c);

    vector_mul<<< tbloco, tthreads >>>(d_a, d_b, d_c);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
    printf("c[0] = %d\n",c[0]);
    printf("c[%d] = %d\n", (NLINHAS*NCOLUNAS)-1, c[(NLINHAS*NCOLUNAS)-1]);

    free(a); free(b); free(c);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
