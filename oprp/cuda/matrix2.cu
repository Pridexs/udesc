
#include <hip/hip_runtime.h>
#include <stdio.h>

#define NLINHAS 1024
#define NCOLUNAS 1024
#define THREADS_PER_BLOCK 1024
//32x32
#define NTHREADS 32
#define NCOLUNASPERBLOCK NCOLUNAS/32
#define NLINHASPERBLOCK NLINHAS/32

__global__ void vector_mul(int *a, int *b, int *c) {
    int i, z, soma = 0;

    int coluna = blockIdx.x * blockDim.x + threadIdx.x;
    int linha =  blockIdx.y * blockDim.y + threadIdx.y;

    __shared__ int s_a[NLINHASPERBLOCK][NCOLUNASPERBLOCK];
    __shared__ int s_b[NLINHASPERBLOCK][NCOLUNASPERBLOCK];

    // Copia Matriz
	for (z = 0; z < blockDim.x; z++) {
        //s_a[threadIdx.x][threadIdx.y] = a[ blockIdx.y * NLINHASPERBLOCK + threadIdx.y ][ z * NCOLUNASPERBLOCK + threadIdx.x ]
        //s_b[threadIdx.x][threadIdx.y] = b[ z * NLINHASPERBLOCK + threadId.x  ][ blockIdx.x * NCOLUNASPERBLOCK + threadIdx.x ]
        
        s_a[threadIdx.x][threadIdx.y] = a[ (NCOLUNAS * (blockIdx.y * NLINHASPERBLOCK + threadIdx.y)) + (z * NCOLUNASPERBLOCK + threadIdx.x) ];
        s_b[threadIdx.x][threadIdx.y] = b[ (NCOLUNAS * (z * NLINHASPERBLOCK + threadIdx.x)) + blockIdx.x * NCOLUNASPERBLOCK + threadIdx.x ];
        
        __syncthreads();
        
		for (i = 0; i < NLINHASPERBLOCK; i++) {
	    	soma += s_a[threadIdx.y][i] * s_b[i][threadIdx.x];
	    }
	}

    c[(NLINHAS * linha) + coluna] = soma;
}

int main(){
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int size = NLINHAS * NCOLUNAS * sizeof(int);
    int i;

    hipMalloc((void **) &d_a, size);
    hipMalloc((void **) &d_b, size);
    hipMalloc((void **) &d_c, size);

    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(size);

    for(i = 0; i < NLINHAS*NCOLUNAS; i++){
        a[i] = b[i] = i;
        c[i] = 0;
    }

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    dim3 tthreads = dim3(32, 32, 1);
    dim3 tbloco = dim3(NLINHAS / NLINHASPERBLOCK, NCOLUNAS / NCOLUNASPERBLOCK,1);
    //vector_mul<<< (N + (THREADS_PER_BLOCK-1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>(d_a, d_b, d_c);

    vector_mul<<< tbloco, tthreads >>>(d_a, d_b, d_c);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
    printf("c[0] = %d\n",c[0]);
    printf("c[%d] = %d\n", (NLINHAS*NCOLUNAS)-1, c[(NLINHAS*NCOLUNAS)-1]);

    free(a); free(b); free(c);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
